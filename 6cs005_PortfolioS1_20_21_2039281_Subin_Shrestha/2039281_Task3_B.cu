#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

__global__ void gaussian_red(unsigned char* img_in, unsigned char* img_out){
    int a = threadIdx.x; //height
    int b = blockIdx.x; //width
    int c = blockDim.x; //dimentions
    int count = blockDim.x * threadIdx.x + blockIdx.x;

    int weight_mat[9] = {1,2,1,2,4,2,1,2,1};
    int sum;
    
    signed int num[9][2];
    num[0][0] = a-1;
    num[0][1] = b-1;
    num[1][0] = a-1;
    num[1][1] = b;
    num[2][0] = a-1;
    num[2][1] = b+1;
    num[3][0] = a;
    num[3][1] = b-1;
    num[4][0] = a;
    num[4][1] = b;
    num[5][0] = a;
    num[5][1] = b+1;
    num[6][0] = a+1;
    num[6][1] = b-1;
    num[7][0] = a+1;
    num[7][1] = b;
    num[8][0] = a+1;
    num[8][1] = b+1;

    int mat[9];
    for(int i =0;i<9;i++){
        if(num[i][0] < 0 || num[i][1] < 0 || num[i][0] > 3|| num[i][1]>3){
            mat[i] = -1;
        }
        else{
            mat[i] = num[i][0] + c*num[i][1];
        }
    }
    sum=0;
    for(int i=0;i<9;i++){
        if(mat[i] == -1){
            sum = sum + (0*weight_mat[i]);
        }
        else{
            
            sum = sum + (img_in[mat[i]]*weight_mat[i]);
        }
    }
    sum = sum/16;

    img_out[4*count] = sum;

    //printf("%d ", sum);
    //printf("a: %d, b: %d, c: %d, count: %d \n", a,b,c,count);
}
__global__ void gaussian_green(unsigned char* img_in, unsigned char* img_out){
    int a = threadIdx.x; //height
    int b = blockIdx.x; //width
    int c = blockDim.x; //dimentions
    int count = blockDim.x * threadIdx.x + blockIdx.x;

    int weight_mat[9] = {1,2,1,2,4,2,1,2,1};
    int sum;
    
    signed int num[9][2];
    num[0][0] = a-1;
    num[0][1] = b-1;
    num[1][0] = a-1;
    num[1][1] = b;
    num[2][0] = a-1;
    num[2][1] = b+1;
    num[3][0] = a;
    num[3][1] = b-1;
    num[4][0] = a;
    num[4][1] = b;
    num[5][0] = a;
    num[5][1] = b+1;
    num[6][0] = a+1;
    num[6][1] = b-1;
    num[7][0] = a+1;
    num[7][1] = b;
    num[8][0] = a+1;
    num[8][1] = b+1;

    int mat[9];
    for(int i =0;i<9;i++){
        if(num[i][0] < 0 || num[i][1] < 0 || num[i][0] > 3|| num[i][1]>3){
            mat[i] = -1;
        }
        else{
            mat[i] = num[i][0] + c*num[i][1];
        }
    }
    sum=0;
    for(int i=0;i<9;i++){
        if(mat[i] == -1){
            sum = sum + (0*weight_mat[i]);
        }
        else{
            
            sum = sum + (img_in[mat[i]]*weight_mat[i]);
        }
    }
    sum = sum/16;

    img_out[(4*count)+1] = sum;

    //printf("%d ", sum);
    //printf("a: %d, b: %d, c: %d, count: %d \n", a,b,c,count);
}
__global__ void gaussian_blue(unsigned char* img_in, unsigned char* img_out){
    int a = threadIdx.x; //height
    int b = blockIdx.x; //width
    int c = blockDim.x; //dimentions
    int count = blockDim.x * threadIdx.x + blockIdx.x;

    int weight_mat[9] = {1,2,1,2,4,2,1,2,1};
    int sum;
    
    signed int num[9][2];
    num[0][0] = a-1;
    num[0][1] = b-1;
    num[1][0] = a-1;
    num[1][1] = b;
    num[2][0] = a-1;
    num[2][1] = b+1;
    num[3][0] = a;
    num[3][1] = b-1;
    num[4][0] = a;
    num[4][1] = b;
    num[5][0] = a;
    num[5][1] = b+1;
    num[6][0] = a+1;
    num[6][1] = b-1;
    num[7][0] = a+1;
    num[7][1] = b;
    num[8][0] = a+1;
    num[8][1] = b+1;

    int mat[9];
    for(int i =0;i<9;i++){
        if(num[i][0] < 0 || num[i][1] < 0 || num[i][0] > 3|| num[i][1]>3){
            mat[i] = -1;
        }
        else{
            mat[i] = num[i][0] + c*num[i][1];
        }
    }
    sum=0;
    for(int i=0;i<9;i++){
        if(mat[i] == -1){
            sum = sum + (0*weight_mat[i]);
        }
        else{
            
            sum = sum + (img_in[mat[i]]*weight_mat[i]);
        }
    }
    sum = sum/16;

    img_out[(4*count)+2] = sum;

    //printf("%d ", sum);
    //printf("a: %d, b: %d, c: %d, count: %d \n", a,b,c,count);
}
__global__ void gaussian_transparent(unsigned char* img_in, unsigned char* img_out){
    int a = threadIdx.x; //height
    int b = blockIdx.x; //width
    int c = blockDim.x; //dimentions
    int count = blockDim.x * threadIdx.x + blockIdx.x;

    int weight_mat[9] = {1,2,1,2,4,2,1,2,1};
    int sum;
    
    signed int num[9][2];
    num[0][0] = a-1;
    num[0][1] = b-1;
    num[1][0] = a-1;
    num[1][1] = b;
    num[2][0] = a-1;
    num[2][1] = b+1;
    num[3][0] = a;
    num[3][1] = b-1;
    num[4][0] = a;
    num[4][1] = b;
    num[5][0] = a;
    num[5][1] = b+1;
    num[6][0] = a+1;
    num[6][1] = b-1;
    num[7][0] = a+1;
    num[7][1] = b;
    num[8][0] = a+1;
    num[8][1] = b+1;

    int mat[9];
    for(int i =0;i<9;i++){
        if(num[i][0] < 0 || num[i][1] < 0 || num[i][0] > 3|| num[i][1]>3){
            mat[i] = -1;
        }
        else{
            mat[i] = num[i][0] + c*num[i][1];
        }
    }
    sum=0;
    for(int i=0;i<9;i++){
        if(mat[i] == -1){
            sum = sum + (0*weight_mat[i]);
        }
        else{
            
            sum = sum + (img_in[mat[i]]*weight_mat[i]);
        }
    }
    sum = sum/16;

    img_out[(4*count)+3] = sum;

    //printf("%d ", sum);
    //printf("a: %d, b: %d, c: %d, count: %d \n", a,b,c,count);
}
int main(int argc, char **argv){

  hipError_t hipError_t;
	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "4x4.png";
  const char* newFileName = "generated.png";

  
	error = lodepng_decode32_file(&image, &width, &height, filename);

  unsigned char r[height*width];
	unsigned char g[height*width];
	unsigned char b[height*width];
	unsigned char t[height*width]; //transparency
	
  if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
  }

   printf("width = %d height = %d\n", width, height);
    for(int i = 0; i<height*width*4; i=i+4){
      r[i/4] = image[i];
      g[i/4] = image[1+i];
      b[i/4] = image[2+i];
      t[i/4] = image[3+i];
    }

  printf("\n");
  const int ARRAY_BYTES = height*width * sizeof(unsigned char);
  unsigned char h_output[height*width*4];


  // declare GPU memory pointers for rgbt arrays
  unsigned char * d_red;
  unsigned char * d_green;
  unsigned char * d_blue;
  unsigned char * d_transparent;

  unsigned char * d_output;

  // allocate GPU memory For rgbt arrays
  hipMalloc((void**) &d_red, ARRAY_BYTES);
  hipMalloc((void**) &d_green, ARRAY_BYTES);
  hipMalloc((void**) &d_blue, ARRAY_BYTES);
  hipMalloc((void**) &d_transparent, ARRAY_BYTES);

  hipMalloc((void**) &d_output, ARRAY_BYTES);

  //Memory copy
  hipMemcpy(d_red, r, ARRAY_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_green, g, ARRAY_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_blue, b, ARRAY_BYTES, hipMemcpyHostToDevice);
  hipMemcpy(d_transparent, t, ARRAY_BYTES, hipMemcpyHostToDevice);

  //<<<blockid.x,threadid.x >>>
  gaussian_red<<<width, height>>>(d_red, d_output);
  gaussian_red<<<width, height>>>(d_green, d_output);
  gaussian_red<<<width, height>>>(d_blue, d_output);
  gaussian_red<<<width, height>>>(d_transparent, d_output);

  hipDeviceSynchronize();
  // copy back the result array to the CPU
  hipMemcpy(h_output, d_output, ARRAY_BYTES, hipMemcpyDeviceToHost);
  
  for(int i = 0; i<height*width*4; i=i+4){
      printf("%d %d %d %d\n", h_output[i], h_output[i+1], h_output[i+2], h_output[i+3]);
  }
  hipError_t = hipGetLastError();
  
  if(hipError_t){
    fprintf(stderr, "Kernel launch returned %d %s\n", 
    hipError_t, hipGetErrorString(hipError_t));
    return 1;
  } else {
    fprintf(stderr, "Kernel launch successful.\n");
  }
  encError = lodepng_encode32_file(newFileName, h_output, width, height);
	if(encError){
		printf("error %u: %s\n", error, lodepng_error_text(encError));
	}
       
      
	return 0;
}