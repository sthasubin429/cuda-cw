#include "hip/hip_runtime.h"
//nvcc 2039281_Task3_B.cu lodepng.cpp -o task3_B 
//task3_B hck.png
#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
__device__ unsigned int width;
__device__ unsigned int height;
__device__ unsigned char getRed(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4);
  return image[i];
}

__device__ unsigned char getGreen(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +1;
  return image[i];
}

__device__ unsigned char getBlue(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +2;
  return image[i];
}

__device__ unsigned char getAlpha(unsigned char *image, unsigned int row, unsigned int col){
  unsigned int i = (row * width * 4) + (col * 4) +3;
  return image[i];
}

__device__ void setRed(unsigned char *image, unsigned int row, unsigned int col, unsigned char red){
  unsigned int i = (row * width * 4) + (col * 4);
  image[i] = red;
}

__device__ void setGreen(unsigned char *image, unsigned int row, unsigned int col, unsigned char green){
  unsigned int i = (row * width * 4) + (col * 4) +1;
  image[i] = green;
}

__device__ void setBlue(unsigned char *image, unsigned int row, unsigned int col, unsigned char blue){
  unsigned int i = (row * width * 4) + (col * 4) +2;
  image[i] = blue;
}

__device__ void setAlpha(unsigned char *image, unsigned int row, unsigned int col, unsigned char alpha){
  unsigned int i = (row * width * 4) + (col * 4) +3;
  image[i] = alpha;
}

__global__ void cudaBlur(unsigned char* img_in, unsigned char* newImage){
    if(blockIdx.x == 0 || threadIdx.x == 0 ){

    }
    else{
        float filter[3][3] = {
  { 1.0/16, 2.0/16, 1.0/16 },
  { 2.0/16, 4.0/16, 2.0/16 },
  { 1.0/16, 2.0/16, 1.0/16 }};

    unsigned redTL,redTC, redTR;
    unsigned redL, redC, redR;
    unsigned redBL,redBC, redBR;
    unsigned newRed;

    unsigned greenTL,greenTC, greenTR;
    unsigned greenL, greenC, greenR;
    unsigned greenBL,greenBC, greenBR;
    unsigned newGreen;

    unsigned blueTL,blueTC, blueTR;
    unsigned blueL, blueC, blueR;
    unsigned blueBL,blueBC, blueBR;
    unsigned newBlue;

    int row = threadIdx.x;
    int col = blockIdx.x;

      setGreen(newImage, row, col, getGreen(img_in, row, col));
      setBlue(newImage, row, col, getBlue(img_in, row, col));
      setAlpha(newImage, row, col, 255);		

      redTL = getRed(img_in, row-1, col-1);
      redTC = getRed(img_in, row-1, col);
      redTR = getRed(img_in, row-1, col+1);

      redL = getRed(img_in, row, col-1);
      redC = getRed(img_in, row, col);
      redR = getRed(img_in, row, col+1);

      redBL = getRed(img_in, row+1, col-1);
      redBC = getRed(img_in, row+1, col);
      redBR = getRed(img_in, row+1, col+1);

      newRed = redTL*filter[0][0] + redTC*filter[0][1] + redTR*filter[0][2]
             + redL*filter[1][0]  + redC*filter[1][1]  + redR*filter[1][2]
             + redBL*filter[2][0] + redBC*filter[2][1] + redBR*filter[2][2];
 
      setRed(newImage, row, col, newRed);

      greenTL = getGreen(img_in, row-1, col-1);
      greenTC = getGreen(img_in, row-1, col);
      greenTR = getGreen(img_in, row-1, col+1);

      greenL = getGreen(img_in, row, col-1);
      greenC = getGreen(img_in, row, col);
      greenR = getGreen(img_in, row, col+1);

      greenBL = getGreen(img_in, row+1, col-1);
      greenBC = getGreen(img_in, row+1, col);
      greenBR = getGreen(img_in, row+1, col+1);

      newGreen = greenTL*filter[0][0] + greenTC*filter[0][1] + greenTR*filter[0][2]
             + greenL*filter[1][0]  + greenC*filter[1][1]  + greenR*filter[1][2]
             + greenBL*filter[2][0] + greenBC*filter[2][1] + greenBR*filter[2][2];
 
      setGreen(newImage, row, col, newGreen);

      blueTL = getBlue(img_in, row-1, col-1);
      blueTC = getBlue(img_in, row-1, col);
      blueTR = getBlue(img_in, row-1, col+1);

      blueL = getBlue(img_in, row, col-1);
      blueC = getBlue(img_in, row, col);
      blueR = getBlue(img_in, row, col+1);

      blueBL = getBlue(img_in, row+1, col-1);
      blueBC = getBlue(img_in, row+1, col);
      blueBR = getBlue(img_in, row+1, col+1);

      newBlue = blueTL*filter[0][0] + blueTC*filter[0][1] + blueTR*filter[0][2]
             + blueL*filter[1][0]  + blueC*filter[1][1]  + blueR*filter[1][2]
             + blueBL*filter[2][0] + blueBC*filter[2][1] + blueBR*filter[2][2];
 
      setBlue(newImage, row, col, newBlue);        
    }
    
}

__global__ void setDimentions(unsigned int givenWidth, unsigned int givenHeight){
    width = givenWidth;
    height = givenHeight;
    
}

int main(int argc, char **argv){
    unsigned char *image;
    unsigned int w;
    unsigned int h;
    const char* filename = argv[1];
    const char* img_output = "generated.png";
    unsigned char* output_raw;
    hipError_t error;
    
    lodepng_decode32_file(&image, &w, &h, filename);
    
    printf("width = %d height = %d\n", w, h);

    setDimentions<<<1,1>>>(w, h);

    unsigned char * d_image;
    const int ARRAY_BYTES = h*w*4 * sizeof(unsigned char);

    output_raw = (unsigned char *)malloc(ARRAY_BYTES);

    hipMalloc((void**) &d_image, ARRAY_BYTES);
    hipMemcpy(d_image, image, ARRAY_BYTES, hipMemcpyHostToDevice);

    unsigned char * d_out;
    hipMalloc((void**) &d_out, ARRAY_BYTES);

    cudaBlur<<<w-1, h-1>>>(d_image, d_out);
    //Declaration of cuda thread synchronize
    error = hipGetLastError();
    if(error){
      fprintf(stderr, "Kernel launch returned %d %s\n", error,
        hipGetErrorString(error));
      exit(1);
    }
    hipDeviceSynchronize();

    hipMemcpy(output_raw, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    
    lodepng_encode32_file(img_output, output_raw, w, h);

    free(image);

    return 0;
}