#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

__global__ void calc(unsigned char* img, int* height, int* width){
    int a = threadIdx.x; //height
    int b = blockIdx.x; //width
    int c = blockDim.x; //dimentions
    int count = blockDim.x * threadIdx.x + blockIdx.x;
    
    printf("a: %d, b: %d, c: %d, count: %d ,height: %d, width: %d \n", a,b,c,count, height,width);
}

int main(int argc, char **argv){

  hipError_t hipError_t;
	unsigned int error;
	//unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	const char* filename = "hck.png";
  //const char* newFileName = "generated.png";

  
	error = lodepng_decode32_file(&image, &width, &height, filename);

  unsigned char r[height*width];
	unsigned char g[height*width];
	unsigned char b[height*width];
	unsigned char t[height*width]; //transparency
	
  if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
  }

   printf("width = %d height = %d\n", width, height);
    for(int i = 0; i<height*width*4; i=i+4){
      r[i/4] = image[i];
      g[i/4] = image[1+i];
      b[i/4] = image[2+i];
      t[i/4] = image[3+i];
    }
  const int ARRAY_BYTES = height*width * sizeof(unsigned char);

  //declare GPU memopry height and width
  unsigned int * d_height, d_width;
  //allocate GPU memory for height and width
  hipMalloc((void**) &d_height, sizeof(int));
  hipMalloc((void**) &d_width, sizeof(int));
  //Memory copy for height and weidth
  hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);

  // declare GPU memory pointers for rgbt arrays
  unsigned char * d_red;
  unsigned char * d_red_out;

  // allocate GPU memory For rgbt arrays
  hipMalloc((void**) &d_red, ARRAY_BYTES);
  hipMalloc((void**) &d_red_out, ARRAY_BYTES);

  //Memory copy
  hipMemcpy(d_red, r, ARRAY_BYTES, hipMemcpyHostToDevice);
  //<<<blockid.x,threadid.x >>>
  calc<<<width, height>>>(d_red);
  //calc<<<1, 1>>>(d_red, d_height, d_width);

  hipDeviceSynchronize();
  hipError_t = hipGetLastError();
  
  if(hipError_t){
    fprintf(stderr, "Kernel launch returned %d %s\n", 
    hipError_t, hipGetErrorString(hipError_t));
    return 1;
  } else {
    fprintf(stderr, "Kernel launch successful.\n");
  }     
      
	return 0;
}